#include "hip/hip_runtime.h"
#include "../../include/parallel.cuh"
// Parallel SpMV 
__global__ 
void spmv_csr(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, const float *x, float *y) {
    // grid stride loop for dot product
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_rows; i += blockDim.x * gridDim.x) {
        float dp = 0;
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        
        for (int j = row_start; j < row_end; ++j) {
            dp += values[j] * x[col_ind[j]];
        }
        
        y[i] = dp;
    }
 }