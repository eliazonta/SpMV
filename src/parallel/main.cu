#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "../../include/parser.h"

int main(int argc, const char * argv[]) {
    fprintf(stdout, "============================\n");
    fprintf(stdout, "*** Parallel SpMV (CUDA) ***\n");
    fprintf(stdout, "----------------------------\n");
    fprintf(stdout, "**** Author: Zonta Elia ****\n");
    fprintf(stdout, "----------------------------\n");
    fprintf(stdout, "============================\n");
    if (argc != 5) {
        fprintf(stdout, "Usage : \n <./bin/SpVM-CUDA> <threads num> <num of iterations> <print mode [1 default, 2 view data info]> <file path>\n");
        exit(-1);
    }
    
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals, numSMs;
    float *values;
    
    int num_thread = atoi(argv[1]);
    int num_repeat = atoi(argv[2]);
    int print_mode = atoi(argv[3]);
    const char *filename = argv[4];
    
    read_matrix(&row_ptr, &col_ind, &values, filename, &num_rows, &num_cols, &num_vals);
    
    float *x = (float *) malloc(num_rows * sizeof(float));
    float *y = (float *) malloc(num_rows * sizeof(float));
    for (int i = 0; i < num_rows; ++i) {
        x[i] = 1.0;
        y[i] = 0.0;
    }
    
    if (print_mode == 2) {
        // Values Array
        fprintf(stdout, "Values Array:\n");
        for (int i = 0; i < num_vals; i++) {
            fprintf(stdout, "%.6f ", values[i]);
        }
        
        // Column Indices Array
        fprintf(stdout, "\n\nColumn Indices Array:\n");
        for (int i = 0; i < num_vals; i++) {
            fprintf(stdout, "%d ", col_ind[i]);
        }
        
        // Row Pointer Array
        fprintf(stdout, "\n\nRow Pointer Array:\n");
        for (int i = 0; i < (num_rows + 1); i++) {
            fprintf(stdout, "%d ", row_ptr[i]);
        }
        
        fprintf(stdout, "\n\nInitial Vector:\n");
        for (int i = 0; i < num_rows; i++) {
            fprintf(stdout, "%.1f ", x[i]);
        }
        
        fprintf(stdout, "\n\nResulting Vector:\n");
    }
    
    // Allocate on device
    int *d_row_ptr, *d_col_ind;
    float *d_values, *d_x, *d_y;
    hipMalloc((void**)&d_row_ptr, (num_rows + 1) * sizeof(int));
    hipMalloc((void**)&d_col_ind, num_vals * sizeof(int));
    hipMalloc((void**)&d_values, num_vals * sizeof(float));
    hipMalloc((void**)&d_x, num_rows * sizeof(float));
    hipMalloc((void**)&d_y, num_rows * sizeof(float));
    
    // Get number of SMs
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    
    // Copy from host to device
    hipMemcpy(d_row_ptr, row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, col_ind, num_vals * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, num_vals * sizeof(float), hipMemcpyHostToDevice);
    
    // Time the iterations
    float elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    for (int i = 0; i < num_repeat; i++) {
        hipMemcpy(d_x, x, num_rows * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, num_rows * sizeof(float), hipMemcpyHostToDevice);
        
        // Call kernel function
        spmv_csr<<<32 * numSMs, num_thread>>>(d_row_ptr, d_col_ind, d_values, num_rows, d_x, d_y);
        
        // Copy the result to x_{i} at the end of each iteration, and use it in iteration x_{i+1}
        hipMemcpy(y, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost);
        for (int i = 0; i < num_rows; i++) {
            x[i] = y[i];
            y[i] = 0.0;
        }
    }
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    // Print resulting vector
    if (print_mode == 2) {
        for (int i = 0; i < num_rows; i++) {
            fprintf(stdout, "%.6f ", x[i]);
        }
        fprintf(stdout, "\n");
    }
    
    // Print elapsed time
    printf("\nParallel Running time:  %.4f ms\n", elapsed_time);
    printf("Num SMs: %d\n", numSMs);
    
    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
    
    free(row_ptr);
    free(col_ind);
    free(values);
    
    return 0;
}

